/*******************************************************
 文件名：ImagePreProcessGPU_kernels.cu
 作者：sharkls
 描述：GPU加速的图像预处理CUDA核函数
 版本：v1.0
 日期：2025-01-20
 *******************************************************/

#include <hip/hip_runtime.h>

#include <cstdint>

// CUDA内核函数定义
__global__ void resizeKernel(uchar3* src, uchar3* dst, int src_width, int src_height, 
                            int dst_width, int dst_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < dst_width && y < dst_height) {
        float src_x = (float)x * src_width / dst_width;
        float src_y = (float)y * src_height / dst_height;
        
        int src_x0 = (int)src_x;
        int src_y0 = (int)src_y;
        int src_x1 = min(src_x0 + 1, src_width - 1);
        int src_y1 = min(src_y0 + 1, src_height - 1);
        
        float fx = src_x - src_x0;
        float fy = src_y - src_y0;
        
        uchar3 p00 = src[src_y0 * src_width + src_x0];
        uchar3 p01 = src[src_y0 * src_width + src_x1];
        uchar3 p10 = src[src_y1 * src_width + src_x0];
        uchar3 p11 = src[src_y1 * src_width + src_x1];
        
        uchar3 result;
        result.x = (unsigned char)((1 - fx) * (1 - fy) * p00.x + fx * (1 - fy) * p01.x + 
                          (1 - fx) * fy * p10.x + fx * fy * p11.x);
        result.y = (unsigned char)((1 - fx) * (1 - fy) * p00.y + fx * (1 - fy) * p01.y + 
                          (1 - fx) * fy * p10.y + fx * fy * p11.y);
        result.z = (unsigned char)((1 - fx) * (1 - fy) * p00.z + fx * (1 - fy) * p01.z + 
                          (1 - fx) * fy * p10.z + fx * fy * p11.z);
        
        dst[y * dst_width + x] = result;
    }
}

__global__ void normalizeKernel(uchar3* src, float* dst, int width, int height, float scale) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = y * width + x;
        uchar3 pixel = src[idx];
        
        // 注意：输入已经是RGB格式（经过BGR到RGB转换）
        dst[idx * 3 + 0] = pixel.x * scale;  // R
        dst[idx * 3 + 1] = pixel.y * scale;  // G
        dst[idx * 3 + 2] = pixel.z * scale;  // B
    }
}

__global__ void hwcToChwKernel(float* src, float* dst, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int hwc_idx = (y * width + x) * 3;
        int chw_idx = y * width + x;
        
        dst[0 * height * width + chw_idx] = src[hwc_idx + 0];  // R -> C0
        dst[1 * height * width + chw_idx] = src[hwc_idx + 1];  // G -> C1
        dst[2 * height * width + chw_idx] = src[hwc_idx + 2];  // B -> C2
    }
}

__global__ void padImageKernel(float* src, float* dst, int src_width, int src_height, 
                              int dst_width, int dst_height, int pad_top, int pad_left, 
                              float pad_value) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < dst_width && y < dst_height) {
        // 检查是否在填充区域
        if (x < pad_left || x >= pad_left + src_width || 
            y < pad_top || y >= pad_top + src_height) {
            // 填充区域，设置为填充值（3通道）
            int dst_idx = (y * dst_width + x) * 3;
            dst[dst_idx + 0] = pad_value;  // R
            dst[dst_idx + 1] = pad_value;  // G
            dst[dst_idx + 2] = pad_value;  // B
        } else {
            // 图像区域，复制源数据（3通道）
            int src_x = x - pad_left;
            int src_y = y - pad_top;
            int src_idx = (src_y * src_width + src_x) * 3;
            int dst_idx = (y * dst_width + x) * 3;
            dst[dst_idx + 0] = src[src_idx + 0];  // R
            dst[dst_idx + 1] = src[src_idx + 1];  // G
            dst[dst_idx + 2] = src[src_idx + 2];  // B
        }
    }
}

__global__ void bgrToRgbKernel(uchar3* bgr, uchar3* rgb, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = y * width + x;
        uchar3 pixel = bgr[idx];
        
        // BGR to RGB conversion
        rgb[idx].x = pixel.z;  // B -> R
        rgb[idx].y = pixel.y;  // G -> G
        rgb[idx].z = pixel.x;  // R -> B
    }
}

// 批量并行预处理内核函数
__global__ void batchPreprocessKernel(uchar3* src_images, float* dst_images,
                                     int* src_widths, int* src_heights,
                                     int* dst_widths, int* dst_heights,
                                     int* target_widths, int* target_heights,
                                     int* pad_tops, int* pad_lefts,
                                     int batch_size, int max_src_width, int max_src_height,
                                     int max_target_width, int max_target_height) {
    int batch_idx = blockIdx.z;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (batch_idx >= batch_size) return;
    
    int src_width = src_widths[batch_idx];
    int src_height = src_heights[batch_idx];
    int dst_width = dst_widths[batch_idx];
    int dst_height = dst_heights[batch_idx];
    int target_width = target_widths[batch_idx];
    int target_height = target_heights[batch_idx];
    int pad_top = pad_tops[batch_idx];
    int pad_left = pad_lefts[batch_idx];
    
    // 计算源图像偏移
    size_t src_offset = batch_idx * max_src_width * max_src_height;
    uchar3* src = src_images + src_offset;
    
    // 计算目标图像偏移
    size_t dst_offset = batch_idx * max_target_width * max_target_height * 3;
    float* dst = dst_images + dst_offset;
    
    if (x < dst_width && y < dst_height) {
        // 双线性插值resize
        float src_x = (float)x * src_width / dst_width;
        float src_y = (float)y * src_height / dst_height;
        
        int src_x0 = (int)src_x;
        int src_y0 = (int)src_y;
        int src_x1 = min(src_x0 + 1, src_width - 1);
        int src_y1 = min(src_y0 + 1, src_height - 1);
        
        float fx = src_x - src_x0;
        float fy = src_y - src_y0;
        
        uchar3 p00 = src[src_y0 * max_src_width + src_x0];
        uchar3 p01 = src[src_y0 * max_src_width + src_x1];
        uchar3 p10 = src[src_y1 * max_src_width + src_x0];
        uchar3 p11 = src[src_y1 * max_src_width + src_x1];
        
        // BGR to RGB conversion and normalization
        float r = ((1 - fx) * (1 - fy) * p00.z + fx * (1 - fy) * p01.z + 
                   (1 - fx) * fy * p10.z + fx * fy * p11.z) / 255.0f;
        float g = ((1 - fx) * (1 - fy) * p00.y + fx * (1 - fy) * p01.y + 
                   (1 - fx) * fy * p10.y + fx * fy * p11.y) / 255.0f;
        float b = ((1 - fx) * (1 - fy) * p00.x + fx * (1 - fy) * p01.x + 
                   (1 - fx) * fy * p10.x + fx * fy * p11.x) / 255.0f;
        
        // 应用填充
        int target_x = x + pad_left;
        int target_y = y + pad_top;
        
        if (target_x < target_width && target_y < target_height) {
            // HWC to CHW format
            int hwc_idx = (target_y * target_width + target_x) * 3;
            dst[hwc_idx + 0] = r;  // R
            dst[hwc_idx + 1] = g;  // G
            dst[hwc_idx + 2] = b;  // B
        }
    }
}

// 核函数启动器（C接口，供C++调用）
extern "C" {
    void launchResizeKernel(uchar3* src, uchar3* dst, int src_width, int src_height, 
                           int dst_width, int dst_height, hipStream_t stream) {
        dim3 block_size(16, 16);
        dim3 grid_size((dst_width + block_size.x - 1) / block_size.x, 
                      (dst_height + block_size.y - 1) / block_size.y);
        
        resizeKernel<<<grid_size, block_size, 0, stream>>>(
            src, dst, src_width, src_height, dst_width, dst_height);
    }
    
    void launchNormalizeKernel(uchar3* src, float* dst, int width, int height, 
                              float scale, hipStream_t stream) {
        dim3 block_size(16, 16);
        dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                      (height + block_size.y - 1) / block_size.y);
        
        normalizeKernel<<<grid_size, block_size, 0, stream>>>(
            src, dst, width, height, scale);
    }
    
    void launchHWCtoCHWKernel(float* src, float* dst, int width, int height, 
                             hipStream_t stream) {
        dim3 block_size(16, 16);
        dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                      (height + block_size.y - 1) / block_size.y);
        
        hwcToChwKernel<<<grid_size, block_size, 0, stream>>>(
            src, dst, width, height);
    }
    
    void launchPadImageKernel(float* src, float* dst, int src_width, int src_height, 
                             int dst_width, int dst_height, int pad_top, int pad_left, 
                             float pad_value, hipStream_t stream) {
        dim3 block_size(16, 16);
        dim3 grid_size((dst_width + block_size.x - 1) / block_size.x, 
                      (dst_height + block_size.y - 1) / block_size.y);
        
        padImageKernel<<<grid_size, block_size, 0, stream>>>(
            src, dst, src_width, src_height, dst_width, dst_height, 
            pad_top, pad_left, pad_value);
    }
    
    void launchBgrToRgbKernel(uchar3* bgr, uchar3* rgb, int width, int height, 
                             hipStream_t stream) {
        dim3 block_size(16, 16);
        dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                      (height + block_size.y - 1) / block_size.y);
        
        bgrToRgbKernel<<<grid_size, block_size, 0, stream>>>(
            bgr, rgb, width, height);
    }
    
    void launchBatchPreprocessKernel(uchar3* src_images, float* dst_images,
                                    int* src_widths, int* src_heights,
                                    int* dst_widths, int* dst_heights,
                                    int* target_widths, int* target_heights,
                                    int* pad_tops, int* pad_lefts,
                                    int batch_size, int max_src_width, int max_src_height,
                                    int max_target_width, int max_target_height,
                                    hipStream_t stream) {
        dim3 block_size(16, 16);
        dim3 grid_size((max_target_width + block_size.x - 1) / block_size.x, 
                      (max_target_height + block_size.y - 1) / block_size.y,
                      batch_size);
        
        batchPreprocessKernel<<<grid_size, block_size, 0, stream>>>(
            src_images, dst_images, src_widths, src_heights,
            dst_widths, dst_heights, target_widths, target_heights,
            pad_tops, pad_lefts, batch_size, max_src_width, max_src_height,
            max_target_width, max_target_height);
    }
} 